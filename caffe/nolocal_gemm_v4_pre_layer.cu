#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "nolocal_gemm_v4_pre_layer.hpp"
namespace caffe {
	
	
	template <typename Dtype>
	__global__ void nonlocal4_distance_forward(const int nthreads, Dtype* const top_data, const Dtype* const bottom_data,
		const int channel, const int inner_shape, const int len) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int ts = index % inner_shape;
			int tg = (index / inner_shape) % channel;
			int tn = index / inner_shape / channel;
			int ta = ts / len;
			int tb = ts % len;
			Dtype tmp = bottom_data[(tn*channel + tg)*len + ta] - bottom_data[(tn*channel + tg)*len + tb];
			top_data[index] = -tmp*tmp;
		}
	}
	
	template <typename Dtype>
	void NolocalGemmV4PreLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype * data = bottom[0]->gpu_data();
		Dtype * top_data = top[0]->mutable_gpu_data();
		int count = top[0]->count();
		nonlocal4_distance_forward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, top_data, data, channel_, inner_shape_*inner_shape_, inner_shape_);
		
	}
	template <typename Dtype>
	__global__ void nonlocal4_distance_backward(const int nthreads, Dtype* const top_diff, const Dtype* const bottom_data,
		const int channel, const int inner_shape, const int len) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int ts = index % inner_shape;
			int tg = (index / inner_shape) % channel;
			int tn = index / inner_shape / channel;
			int ta = ts / len;
			int tb = ts % len;
			if (ta > tb) {
				Dtype tmp = 2 * (bottom_data[(tn*channel + tg)*len + tb] - bottom_data[(tn*channel + tg)*len + ta]);
				int tid = ((tn*channel + tg)*len + tb)*len + ta;
				tmp = (top_diff[index] + top_diff[tid])*tmp;
				top_diff[index] = tmp;
				top_diff[tid] = -tmp;
			}
			else if (ta == tb) {
				top_diff[index] = 0;
			}

		}
	}
	template <typename Dtype>
	void NolocalGemmV4PreLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		int count = top[0]->count();
		nonlocal4_distance_backward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, top[0]->mutable_gpu_diff(), bottom[0]->gpu_data(), channel_, inner_shape_*inner_shape_, inner_shape_);
		caffe_gpu_set(inner_shape_, Dtype(1.), ones_.mutable_gpu_data());
		caffe_gpu_gemv(CblasNoTrans, num_*channel_*inner_shape_, inner_shape_, Dtype(1.0), top[0]->gpu_diff(),
			ones_.gpu_data(), Dtype(0), bottom[0]->mutable_gpu_diff());
	
	}

	INSTANTIATE_LAYER_GPU_FUNCS(NolocalGemmV4PreLayer);

}  // namespace caffe
