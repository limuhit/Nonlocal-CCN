#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "nolocal_gemm_v4_post_layer.hpp"
namespace caffe {
	
	
	
	template <typename Dtype>
	__global__ void nonlocal4_constrain_forward(const int nthreads, Dtype* const weight, 
		const int inner_shape, const int width, bool code) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index % inner_shape;
			int th = (index / inner_shape) % inner_shape;
			if (code) {
				if (th / width + th%width <= tw / width + tw%width)
					weight[index] = 0;
			}
			else {
				if (th / width + th%width < tw / width + tw%width)
					weight[index] = 0;
			}

		}
	}
	template <typename Dtype>
	__global__ void nonlocal4_post(const int nthreads, Dtype* const data, Dtype* const dist,
		const Dtype mean, const Dtype max_distance, const int inner_shape) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tidx = index*inner_shape;
			data[tidx] = mean;
			dist[tidx] = max_distance;

		}
	}
	template <typename Dtype>
	__global__ void nonlocal4_div_forward(const int nthreads, const Dtype* const input, const Dtype* const sum, Dtype* const output, 
	    const int inner_shape) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int th = (index / inner_shape) % inner_shape;
			int tn = (index / inner_shape) / inner_shape;
			if (sum[tn*inner_shape + th] > 0)
				output[index] = input[index] / sum[tn*inner_shape + th];
			else
				output[index] = input[index];
		}
	}
	template <typename Dtype>
	void NolocalGemmV4PostLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype * bottom_data = bottom[0]->gpu_data();
		const Dtype * bottom_wt = bottom[1]->mutable_gpu_data();
		Dtype * wt = tmp_.mutable_gpu_data();
		Dtype * top_data = top[0]->mutable_gpu_data();
		int count = bottom[1]->count();
		caffe_gpu_exp(count, bottom_wt, wt);
		nonlocal4_constrain_forward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, wt, inner_shape_, width_, code_);
		caffe_gpu_set(inner_shape_, Dtype(1.), ones_.mutable_gpu_data());
		caffe_gpu_gemv(CblasNoTrans, num_*channel_*inner_shape_, inner_shape_, Dtype(1.0), wt, 
			ones_.gpu_data(), Dtype(0), sdata_.mutable_gpu_data());
		nonlocal4_div_forward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, wt, sdata_.gpu_data(),tmp_.mutable_gpu_diff(), inner_shape_);
		caffe_gpu_gemm_batch2(CblasNoTrans, CblasNoTrans, inner_shape_,1, inner_shape_, Dtype(1.0),
			tmp_.gpu_diff(),bottom_data, Dtype(0), top_data, inner_shape_*inner_shape_, inner_shape_,
			inner_shape_, num_*ngroup_);
		caffe_gpu_mul(count, tmp_.gpu_diff(), bottom_wt, bottom[1]->mutable_gpu_diff());
		caffe_gpu_gemv(CblasNoTrans, num_*channel_*inner_shape_, inner_shape_, Dtype(1.0), bottom[1]->gpu_diff(),
			ones_.gpu_data(), Dtype(0), top[1]->mutable_gpu_data());
		count = num_*channel_;
		nonlocal4_post<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, top[0]->mutable_gpu_data(), top[1]->mutable_gpu_data(), mean_, -margin_*margin_, width_*height_);
	}

	
	template <typename Dtype>
	__global__ void nonlocal4_distance_backward(const int nthreads, const Dtype* const top_diff, const Dtype*  const da,
		const Dtype* const db, Dtype* const da_diff, Dtype* const db_diff,
	    const int inner_shape) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int ph = (index / inner_shape) % inner_shape;
			int pn = index / inner_shape / inner_shape;
			int pidx = pn*inner_shape + ph;
			if (db[index] > 0)
				da_diff[index] = log(db[index])*top_diff[pidx];
			else
				da_diff[index] = 0;
			db_diff[index] = da[index] * top_diff[pidx];
		}
	}
	template <typename Dtype>
	__global__ void nonlocal4_softmax_backward(const int nthreads, const Dtype* const top_diff,	const Dtype* const top_data,
		const Dtype * const sdiff,Dtype* const bottom_diff, const int inner_shape) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int ph = (index / inner_shape) % inner_shape;
			int pn = index / inner_shape / inner_shape;
			int pidx = pn*inner_shape + ph;
			bottom_diff[index] = bottom_diff[index] + (top_diff[index] - sdiff[pidx])*top_data[index];
		}
	}
	template <typename Dtype>
	void NolocalGemmV4PostLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		
		const Dtype * top_diff = top[0]->gpu_diff();
		const Dtype * bottom_data = bottom[0]->gpu_data();
		int count = bottom[1]->count();
		caffe_gpu_memcpy(count * sizeof(Dtype), tmp_.gpu_data(), bottom[1]->mutable_gpu_data());
		caffe_gpu_memcpy(count * sizeof(Dtype), tmp_.gpu_diff(), tmp_.mutable_gpu_data());
		nonlocal4_distance_backward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, top[1]->gpu_diff(), tmp_.gpu_data(), bottom[1]->gpu_data(),tmp_.mutable_gpu_diff(),
				bottom[1]->mutable_gpu_diff(), inner_shape_);
		caffe_gpu_gemm_batch2(CblasTrans, CblasNoTrans, inner_shape_, 1, inner_shape_, Dtype(1.0),
			tmp_.gpu_data(), top_diff, Dtype(0), bottom[0]->mutable_gpu_diff(), inner_shape_*inner_shape_, inner_shape_,
			inner_shape_, num_*ngroup_);
		caffe_gpu_gemm_batch2(CblasNoTrans, CblasTrans,  inner_shape_, inner_shape_, 1, Dtype(1.0),
			top_diff, bottom_data, Dtype(1.0), tmp_.mutable_gpu_diff(), inner_shape_, inner_shape_,
			inner_shape_*inner_shape_, num_*ngroup_);
		//caffe_gpu_memcpy(count * sizeof(Dtype), tmp_.gpu_diff(), bottom[1]->mutable_gpu_diff());
		//caffe_gpu_set(count, Dtype(1. / (2 * 4 * 16 * 16)), tmp_.mutable_gpu_diff());
		//caffe_gpu_set(count, Dtype(0), bottom[1]->mutable_gpu_diff());
		
		caffe_gpu_gemm_batch2(CblasNoTrans, CblasNoTrans, 1, 1, inner_shape_, Dtype(1.0),
			tmp_.gpu_data(), tmp_.gpu_diff(), Dtype(0), sdata_.mutable_gpu_diff(), inner_shape_, inner_shape_,
			1, num_*ngroup_*inner_shape_);
		nonlocal4_softmax_backward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, tmp_.gpu_diff(),  tmp_.gpu_data(), sdata_.gpu_diff(),bottom[1]->mutable_gpu_diff(),inner_shape_);
		

	}

	INSTANTIATE_LAYER_GPU_FUNCS(NolocalGemmV4PostLayer);

}  // namespace caffe
